#include "hip/hip_runtime.h"
//
// Created by troy on 19-5-13.
//

#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/focal_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void
FocalLossForwardGPU(const int nthreads, const Dtype *prob_data,
                    const Dtype *label, Dtype *loss, const int num,
                    const int dim, const int spatial_dim,
                    const bool has_ignore_label_, const int ignore_label_,
                    Dtype *counts, float alpha_, float gamma_) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int i = index / spatial_dim; // batch index
    const int s = index % spatial_dim; // index of pred
    const int label_value = static_cast<int>(label[i]);
    if (has_ignore_label_ && label_value == ignore_label_) {
      loss[index] = 0;
      counts[index] = 0;
    } else {
      // loss[index] = -log(max(prob_data[n * dim + label_value * spatial_dim +
      // s],
      //                Dtype(FLT_MIN)));
      Dtype pt = prob_data[i * dim + s];
      loss[index] =
          -alpha_ * pow(1.0 - pt, gamma_) * log(max(pt, Dtype(FLT_MIN)));
      counts[index] = 1;
    }
  }
}

template <typename Dtype>
void FocalLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype> *> &bottom,
                                        const vector<Blob<Dtype> *> &top) {
  const Dtype *input_data = bottom[0]->gpu_data();
  const Dtype *target = bottom[1]->gpu_data();
  const int dim = bottom[0]->count() / outer_num_;
  const int nthreads = outer_num_ * inner_num_;
  // Since this memory is not used for anything until it is overwritten
  // on the backward pass, we use it here to avoid having to allocate new GPU
  // memory to accumulate intermediate results in the kernel.
  Dtype *loss_data = bottom[0]->mutable_gpu_diff();
  // Similarly, this memory is never used elsewhere, and thus we can use it
  // to avoid having to allocate additional GPU memory.
  Dtype *counts = bottom[0]->mutable_gpu_diff();
  // NOLINT_NEXT_LINE(whitespace/operators)
  FocalLossForwardGPU<Dtype>
      <<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
          nthreads, input_data, target, loss_data, outer_num_, dim, inner_num_,
          has_ignore_label_, ignore_label_, counts, alpha_, gamma_);
  Dtype loss;
  caffe_gpu_asum(nthreads, loss_data, &loss);
  Dtype valid_count = -1;
  // Only launch another CUDA kernel if we actually need the count of valid
  // outputs.
  if (normalization_ == LossParameter_NormalizationMode_VALID &&
      has_ignore_label_) {
    caffe_gpu_asum(nthreads, counts, &valid_count);
  }
  Dtype normalizer = this->get_normalizer(normalization_, valid_count);
  top[0]->mutable_cpu_data()[0] = loss / normalizer;
  if (top.size() == 2) {
    top[1]->ShareData(*bottom[0]);
  }
}

template <typename Dtype>
__global__ void
FocalLossBackwardGPU(const int nthreads, const Dtype *top, const Dtype *label,
                     Dtype *bottom_diff, const int num, const int dim,
                     const int spatial_dim, const bool has_ignore_label_,
                     const int ignore_label_, Dtype *counts, float alpha_,
                     float gamma_) {
  const int channels = dim / spatial_dim;

  CUDA_KERNEL_LOOP(index, nthreads) {
    const int i = index / spatial_dim; // batch index
    const int s = index % spatial_dim; // index of prob
    const int label_value = static_cast<int>(label[i]);

    if (has_ignore_label_ && label_value == ignore_label_) {
      for (int c = 0; c < channels; ++c) {
        bottom_diff[i * dim + c] = 0;
      }
    } else {
      Dtype pt = bottom_diff[i * dim + s];
      if (s == label_value) {
        bottom_diff[i * dim + s] =
            alpha_ * gamma_ * log(max(pt, Dtype(FLT_MIN))) *
                pow(1.0 - pt, gamma_ - 1) -
            (alpha_ * pow(1.0 - pt, gamma_)) / max(pt, Dtype(FLT_MIN));
      } else {
        bottom_diff[i * dim + s] =
            -((alpha_ * pow(pt, gamma_)) / (pt - 1.0)) -
            alpha_ * pow(pt, gamma_ - 1) * gamma_ *
                log(max(Dtype(1.0 - pt), Dtype(FLT_MIN)));
      }
      ++(*counts);
    }
  }
}

template <typename Dtype>
void FocalLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype> *> &top,
                                         const vector<bool> &propagate_down,
                                         const vector<Blob<Dtype> *> &bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    Dtype *bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype *prob_data = bottom[0]->gpu_data();
    const Dtype *top_data = top[0]->gpu_data();
    caffe_gpu_memcpy(bottom[0]->count() * sizeof(Dtype), prob_data,
                     bottom_diff);
    const Dtype *label = bottom[1]->gpu_data();
    const int dim = bottom[0]->count() / outer_num_;
    const int nthreads = outer_num_ * inner_num_;
    // Since this memory is never used for anything else,
    // we use to to avoid allocating new GPU memory.
    Dtype valid_counts;
    // NOLINT_NEXT_LINE(whitespace/operators)
    FocalLossBackwardGPU<Dtype>
        <<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
            nthreads, top_data, label, bottom_diff, outer_num_, dim, inner_num_,
            has_ignore_label_, ignore_label_, &valid_counts, alpha_, gamma_);
    // Only launch another CUDA kernel if we actually need the count of valid
    // outputs.
    Dtype normalizer = this->get_normalizer(normalization_, valid_counts);
    const Dtype loss_weight = top[0]->cpu_diff()[0] / normalizer;
    caffe_gpu_scal(bottom[0]->count(), loss_weight, bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(FocalLossLayer);

}