#include "hip/hip_runtime.h"
//
// Created by troy on 19-5-17.
//

#include "caffe/layers/accuracy_generic_layer.hpp"

namespace caffe {

template<typename Dtype>
__global__ void AccuracyPNSRGPU(const int nthreads,
                                const Dtype *bottom_data, const Dtype *label, Dtype *pnsr) {}

template<typename Dtype>
void AccuracyGenericLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype> *> &bottom, const vector<Blob<Dtype> *> &top) {
  const Dtype *bottom_data = bottom[0]->gpu_data();
  const Dtype *bottom_label = bottom[1]->gpu_data();
  switch (this->acc_type) {
  case AccuracyGenericParameter_Type_PSNR:
    const Dtype *mse = bottom[1]->mutable_gpu_diff();
    caffe_gpu_sub(bottom[1]->count(), bottom_data, bottom_label, mse);
    caffe_gpu_powx(bottom[1]->count(), mse, Dtype(2), mse);
    Dtype sum_mse;
    caffe_gpu_asum(bottom[1]->count(), mse, &sum_mse);
    sum_mse /= (bottom[1]->width() * bottom[1]->height() * bottom[1]->num());
    Dtype max;
    if (!this->layer_param().accuracy_generic_param().has_max()) {
      LOG(INFO) << "max not specified, use 255 instead by default";
      max = Dtype(255);
    } else {
      max = Dtype(this->layer_param().accuracy_generic_param().max());
    }
    Dtype pnsr = 20 * log10(max / (sqrt(sum_mse)));
    top[0]->mutable_cpu_data()[0] = pnsr / bottom[1]->num();
    break;
  }
}

template<typename Dtype>
void AccuracyGenericLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype> *> &top,
                                               const vector<bool> &propagate_down,
                                               const vector<Blob<Dtype> *> &bottom) override {
  for (const auto &i: propagate_down) {
    if (i)
      NOT_IMPLEMENTED;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(AccuracyGenericLayer);
} // namespace caffe