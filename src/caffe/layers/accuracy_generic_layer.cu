#include "hip/hip_runtime.h"
//
// Created by troy on 19-5-17.
//

#include "caffe/layers/accuracy_generic_layer.hpp"

namespace caffe {

template<typename Dtype>
__global__ void AccuracyPNSRGPU(const int nthreads,
                                const Dtype *bottom_data, const Dtype *label, Dtype *pnsr) {}

template<typename Dtype>
void AccuracyGenericLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype> *> &bottom, const vector<Blob<Dtype> *> &top) {
  const Dtype *bottom_data = bottom[0]->gpu_data();
  const Dtype *bottom_label = bottom[1]->gpu_data();
  switch (this->acc_type) {
  case AccuracyGenericParameter_Type_PSNR:
    const Dtype *temp = bottom[1]->mutable_gpu_diff();
    Dtype acc;
    caffe_gpu_sub(bottom[1]->count(), bottom_data, bottom_label, temp);
    caffe_gpu_dot(bottom[1]->count(), temp, temp, &acc)
    acc /= bottom[1]->count();

//    Dtype max;
//    if (!this->layer_param().accuracy_generic_param().has_max()) {
//      LOG(INFO) << "max not specified, use 255 instead by default";
//      max = Dtype(255);
//    } else {
//      max = Dtype(this->layer_param().accuracy_generic_param().max());
//    }
    acc = 10 * log10(Dtype(1) / acc);
    top[0]->mutable_cpu_data()[0] = acc;
    break;
  }
}

template<typename Dtype>
void AccuracyGenericLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype> *> &top,
                                               const vector<bool> &propagate_down,
                                               const vector<Blob<Dtype> *> &bottom) override {
  for (const auto &i: propagate_down) {
    if (i)
      NOT_IMPLEMENTED;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(AccuracyGenericLayer);
} // namespace caffe