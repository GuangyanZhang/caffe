#include "hip/hip_runtime.h"
//
// Created by troy on 19-5-17.
//

#include "caffe/layers/accuracy_generic_layer.hpp"

namespace caffe {

template<typename Dtype>
__global__ void AccuracyPNSRGPU(const int nthreads,
                                const Dtype *bottom_data, const Dtype *label, Dtype *pnsr) {}

template<typename Dtype>
void AccuracyGenericLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype> *> &bottom, const vector<Blob<Dtype> *> &top) {
  const Dtype *bottom_data = bottom[0]->gpu_data();
  const Dtype *bottom_label = bottom[1]->gpu_data();
  switch (this->acc_type) {
  case AccuracyGenericParameter_Type_PSNR:Blob<Dtype> mse;
    mse.CopyFrom(*bottom[0], false, true);
    caffe_gpu_sub(mse.count(), bottom_data, bottom_label, mse.mutable_gpu_data());
    caffe_gpu_powx(mse.count(), mse.gpu_data(), Dtype(2), mse.mutable_gpu_data());
    Dtype sum_mse;
    caffe_gpu_asum(mse.count(), mse.cpu_data(), &sum_mse);
    sum_mse /= (mse.width() * mse.height() * mse.num());
    Dtype max;
    if (!this->layer_param().accuracy_generic_param().has_max()) {
      LOG(WARNING) << "max not specified, use 255 instead by default";
      max = Dtype(255);
    } else {
      max = Dtype(this->layer_param().accuracy_generic_param().max());
    }
    Dtype pnsr = 20 * log(10 * (max / (sqrt(sum_mse))));
    top[0]->mutable_cpu_data()[0] = pnsr / mse.num();
    break;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(AccuracyGenericLayer);
} // namespace caffe